PIC_LD=ld

ARCHIVE_OBJS=

VCS_ARC0 =_csrc0.so

VCS_OBJS0 =objs/reYIK_d.o objs/g7hgQ_d.o amcQwB.o 



%.o: %.c
	$(CC_CG) $(CFLAGS_CG) -c -o $@ $<

$(VCS_ARC0) : $(VCS_OBJS0)
	$(PIC_LD) -shared  -o .//../simv.daidir//$(VCS_ARC0) $(VCS_OBJS0)
	rm -f $(VCS_ARC0)
	@ln -sf .//../simv.daidir//$(VCS_ARC0) $(VCS_ARC0)

CU_UDP_OBJS = \


CU_LVL_OBJS = \
SIM_l.o 

MAIN_OBJS = \
objs/amcQw_d.o 

CU_OBJS = $(MAIN_OBJS) $(ARCHIVE_OBJS) $(VCS_ARC0) $(CU_UDP_OBJS) $(CU_LVL_OBJS)

